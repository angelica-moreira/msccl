#include "hip/hip_runtime.h"
#include "synchronize.h"
#include <stdio.h>

#define SCCL_MAX_ITER 65536

// flags are a 3-tuple of (workindex, gridoffset_iter, step) and it follows a lexicographical order. a threadblock is ahead of another iff its flag is ahead 
#define COMPUTE_FLAG(__WORKINDEX__,__GRIDOFFSET_ITER__,__STEP__) \
  SCCL_MAX_ITER*SCCL_MAX_NUM_STEPS*(uint64_t)__WORKINDEX__ + ((uint64_t)__GRIDOFFSET_ITER__ * SCCL_MAX_NUM_STEPS + (uint64_t)__STEP__)

__global__ void scclSynchronize(int workIndex, struct ncclDevComm* comm) {
    int tid = threadIdx.x;
    volatile struct scclFlag* scclFlags = comm->scclAlgoShared.flags;
    uint64_t curFlag = COMPUTE_FLAG(workIndex, 0, 0);
    scclFlags[tid].flag = curFlag;
    uint64_t goalFlag = COMPUTE_FLAG(workIndex, 0, 3);
    while ((scclFlags + tid)->flag < goalFlag){};
}